#include "hip/hip_runtime.h"
/*
__device__ const double EPSILON = 1e-5;
__device__ const double BOUNDARY = 1.0 - EPSILON;
*/
__device__ double clamp(double x, double min, double max) {
    if (x < min) {
        return min;
    }
    if (x > max) {
        return max;
    }

    return x;
}

__device__ double distance(double u0, double u1, double v0, double v1) {
    double EPSILON = 1e-5;
    double BOUNDARY = 1.0 - EPSILON;
    if ((fabs(u0 - v0) <= EPSILON) && (fabs(u1 - v1) <= EPSILON)) {
        return 0.0;
    }

    double uv2 = ((u0 - v0) * (u0 - v0)) + ((u1 - v1) * (u1 - v1));
    double u_sq = clamp(u0 * u0 + u1 * u1, 0, BOUNDARY);
    double v_sq = clamp(v0 * v0 + v1 * v1, 0, BOUNDARY);
    double alpha = 1.0 - u_sq;
    double beta = 1.0 - v_sq;
    double result = acosh( 1.0 + 2.0 * uv2 / ( alpha * beta ) );

    return result;
}

__device__ double distance_grad(double u0, double u1, double v0, double v1, int ax) {
    double EPSILON = 1e-5;
    double BOUNDARY = 1.0 - EPSILON;
    if ((fabs(u0 - v0) <= EPSILON) && (fabs(u1 - v1) <= EPSILON)) {
        return 0.0;
    }

    double MACHINE_EPSILON = 2.220446049250313e-16;

    double a = u0 - v0;
    double b = u1 - v1;
    double uv2 = a * a + b * b;

    double u_sq = clamp(u0 * u0 + u1 * u1, 0, BOUNDARY);
    double v_sq = clamp(v0 * v0 + v1 * v1, 0, BOUNDARY);
    double alpha = 1.0 - u_sq;
    double beta = 1.0 - v_sq;

    double gamma = 1.0 + (2.0 / (alpha * beta)) * uv2;
    double shared_scalar = 4.0 / fmax(beta * sqrt((gamma * gamma) - 1.0), MACHINE_EPSILON);

    double u_scalar = (v_sq - 2.0 * (u0 * v0 + u1 * v1) + 1.0) / (alpha * alpha);
    double v_scalar = 1.0 / alpha;

    if (ax == 0) {
        return shared_scalar * (u_scalar * u0 - v_scalar * v0);
    }

    return shared_scalar * (u_scalar * u1 - v_scalar * v1);
}

__device__ void negative_gradient(int i, int n_dimensions, double *pos, double *neg_f, double *sumQs) {

    double qij = 0.0;
    double dij = 0.0;
    double dij_sq = 0.0;

    for (int j = start; j < n_samples; j++) {
        if (i == j) {
            continue;
        }

        dij = distance(pos[i*2 + 0], pos[i*2 + 1], pos[j*2 + 0], pos[j*2 + 1]);
        dij_sq = dij * dij;

        qij = 1.0 / (1.0 + dij_sq);

        double mult = qij * qij;

        /*
        if (true) {
            // New Fix
            mult = qij * qij * dij;
        }
        else {
            // Old Solution
            mult = qij * qij;
        }*/

        sumQs[i] += qij;
        for (int ax = 0; ax < n_dimensions; ax++) {
            neg_f[i * n_dimensions + ax] += mult * distance_grad(pos[i*2 + 0], pos[i*2 + 1], pos[j*2 + 0], pos[j*2 + 1], ax);
            //neg_f[i * n_dimensions + ax] = distance_grad(pos[i*2 + 0], pos[i*2 + 1], pos[j*2 + 0], pos[j*2 + 1], ax);
            //neg_f[i * n_dimensions + ax] = mult;
            //neg_f[i * n_dimensions + ax] = distance(0.1, -0.1, 0.3, 0.5);
            //neg_f[i * n_dimensions + ax] = distance_grad(0.1, -0.1, 0.3, 0.5, 0);
        }
    }
}

__device__ void positive_gradient() {
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i > n_samples) {
        return;
    }

    double qij = 0.0;
    double dij = 0.0;
    double dij_sq = 0.0;

    for (int j = start; j < n_samples; j++) {
        if (i == j) {
            continue;
        }

        dij = distance(pos[i*2 + 0], pos[i*2 + 1], pos[j*2 + 0], pos[j*2 + 1]);
        dij_sq = dij * dij;

        qij = 1.0 / (1.0 + dij_sq);

        double mult = qij * qij;

        /*
        if (true) {
            // New Fix
            mult = qij * qij * dij;
        }
        else {
            // Old Solution
            mult = qij * qij;
        }*/

        sumQs[i] += qij;
        for (int ax = 0; ax < n_dimensions; ax++) {
            neg_f[i * n_dimensions + ax] += mult * distance_grad(pos[i*2 + 0], pos[i*2 + 1], pos[j*2 + 0], pos[j*2 + 1], ax);
            //neg_f[i * n_dimensions + ax] = distance_grad(pos[i*2 + 0], pos[i*2 + 1], pos[j*2 + 0], pos[j*2 + 1], ax);
            //neg_f[i * n_dimensions + ax] = mult;
            //neg_f[i * n_dimensions + ax] = distance(0.1, -0.1, 0.3, 0.5);
            //neg_f[i * n_dimensions + ax] = distance_grad(0.1, -0.1, 0.3, 0.5, 0);
        }
    }

    /*
    for i in prange(start, n_samples, schedule='static'):
        # Init the gradient vector
        for ax in range(n_dimensions):
            pos_f[i * n_dimensions + ax] = 0.0
        # Compute the positive interaction for the nearest neighbors
        for k in range(indptr[i], indptr[i+1]):
            j = neighbors[k]
            pij = val_P[k]

            dij = distance(pos_reference[i, 0], pos_reference[i, 1], pos_reference[j, 0], pos_reference[j, 1])
            dij_sq = dij * dij

            qij = 1. / (1. + dij_sq)

            if GRAD_FIX:
                # New Fix
                mult = pij * qij * dij
            else:
                # Old solution
                mult = pij * qij

            # only compute the error when needed
            if compute_error:
                qij = qij / sum_Q
                C += pij * log(max(pij, FLOAT32_TINY) / max(qij, FLOAT32_TINY))
            for ax in range(n_dimensions):
                pos_f[i * n_dimensions + ax] += mult * distance_grad(pos_reference[i, 0], pos_reference[i, 1], pos_reference[j, 0], pos_reference[j, 1], ax)
    
    */
}

__global__ void gradient(int start, int n_samples, int n_dimensions, double *pos, double *neg_f, double *sumQs) {
    int i = threadIdx.x + blockIdx.x * blockDim.x + start;

    // i = [start, n_samples)
    if (i > n_samples) {
        return;
    }
}