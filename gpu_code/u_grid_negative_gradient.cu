
#include <hip/hip_runtime.h>
/*
__device__ const double EPSILON = 1e-5;
__device__ const double BOUNDARY = 1.0 - EPSILON;
*/
__device__ double clamp(double x, double min, double max) {
    if (x < min) {
        return min;
    }
    if (x > max) {
        return max;
    }

    return x;
}

__device__ double distance(double u0, double u1, double v0, double v1) {
    double EPSILON = 1e-5;
    double BOUNDARY = 1.0 - EPSILON;
    if ((fabs(u0 - v0) <= EPSILON) && (fabs(u1 - v1) <= EPSILON)) {
        return 0.0;
    }

    double uv2 = ((u0 - v0) * (u0 - v0)) + ((u1 - v1) * (u1 - v1));
    double u_sq = clamp(u0 * u0 + u1 * u1, 0, BOUNDARY);
    double v_sq = clamp(v0 * v0 + v1 * v1, 0, BOUNDARY);
    double alpha = 1.0 - u_sq;
    double beta = 1.0 - v_sq;
    double result = acosh( 1.0 + 2.0 * uv2 / ( alpha * beta ) );

    return result;
}

__device__ double distance_grad(double u0, double u1, double v0, double v1, int ax) {
    double EPSILON = 1e-5;
    double BOUNDARY = 1.0 - EPSILON;
    if ((fabs(u0 - v0) <= EPSILON) && (fabs(u1 - v1) <= EPSILON)) {
        return 0.0;
    }

    double MACHINE_EPSILON = 2.220446049250313e-16;

    double a = u0 - v0;
    double b = u1 - v1;
    double uv2 = a * a + b * b;

    double u_sq = clamp(u0 * u0 + u1 * u1, 0, BOUNDARY);
    double v_sq = clamp(v0 * v0 + v1 * v1, 0, BOUNDARY);
    double alpha = 1.0 - u_sq;
    double beta = 1.0 - v_sq;

    double gamma = 1.0 + (2.0 / (alpha * beta)) * uv2;
    double shared_scalar = 4.0 / fmax(beta * sqrt((gamma * gamma) - 1.0), MACHINE_EPSILON);

    double u_scalar = (v_sq - 2.0 * (u0 * v0 + u1 * v1) + 1.0) / (alpha * alpha);
    double v_scalar = 1.0 / alpha;

    if (ax == 0) {
        return shared_scalar * (u_scalar * u0 - v_scalar * v0);
    }

    return shared_scalar * (u_scalar * u1 - v_scalar * v1);
}

__global__ void add(int start,
                    int n_samples,
                    int n_dimensions,
                    int grid_size,
                    int grid_n,
                    double *pos,
                    double *neg_f,
                    int* grid_square_indices_per_point,
                    int* result_indices, 
                    int* result_starts_counts,
                    double* max_distances,
                    double* square_positions,
                    double *sumQ) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    /*
    for (int k = 0; k < n_samples; k++) {
        neg_f[k] = 1.0;
    }
    atomicAdd(sumQ, 1.0);

    return;
    */
    if (i > n_samples) {
        return;
    }

    int i_grid_index = grid_square_indices_per_point[i];
    int grid_x = i_grid_index % grid_n;
    int grid_y = int(i_grid_index / grid_n);

    double qij = 0.0;
    double dij = 0.0;
    double dij_sq = 0.0;
    double thread_sQ = 0.0;
    double max_square_dist = 0.0;
    double dist_to_square = 0.0;
    double theta = 0.5;
    double theta_sq = theta*theta;
    int point_count = 0;
    int range = 1;

    // Looping over all grid squares
    for (int k = 0; k < grid_size; k++) {
        //for (int k = i_grid_index; k <= i_grid_index; k++) {
        point_count = result_starts_counts[k*2+1];

        // Check for empty square
        if (point_count == 0) {
            continue;
        }

        int k_grid_x = k % grid_n;
        int k_grid_y = int(k / grid_n);

        //max_square_dist = max_distances[k];
        //dist_to_square = distance(pos[i*2 + 0], pos[i*2 + 1], square_positions[k*2 + 0], square_positions[k*2 + 1]);

        // If the square error is relatively small, compute using only the square average
        //if ((max_square_dist*max_square_dist) / dist_to_square < theta_sq) {
        
        if (true){//i_grid_index != k) {//true || (abs(k_grid_x - grid_x) > range || abs(k_grid_y - grid_y) > range)) {
            dij = distance(pos[i*2 + 0], pos[i*2 + 1], square_positions[k*2 + 0], square_positions[k*2 + 1]);
            dij_sq = dij * dij;

            qij = 1.0 / (1.0 + dij_sq);

            double mult = qij * qij;

            thread_sQ += qij * point_count;
            for (int ax = 0; ax < n_dimensions; ax++) {
                neg_f[i * n_dimensions + ax] += point_count * mult * distance_grad(pos[i*2 + 0], pos[i*2 + 1], square_positions[k*2 + 0], square_positions[k*2 + 1], ax);
                //neg_f[i * n_dimensions + ax] = distance_grad(pos[i*2 + 0], pos[i*2 + 1], pos[j*2 + 0], pos[j*2 + 1], ax);
                //neg_f[i * n_dimensions + ax] = mult;
                //neg_f[i * n_dimensions + ax] = distance(0.1, -0.1, 0.3, 0.5);
                //neg_f[i * n_dimensions + ax] = distance_grad(0.1, -0.1, 0.3, 0.5, 0);
            }
            
            // Move on to next grid square
            continue;
        }

        // Otherwise check all points in the square
        for (int m = result_starts_counts[k*2+0]; m < result_starts_counts[k*2+0] + point_count; m++) {
            int j = result_indices[m];
            if (j == i) {
                continue;
            }
            
            dij = distance(pos[i*2 + 0], pos[i*2 + 1], pos[j*2 + 0], pos[j*2 + 1]);
            dij_sq = dij * dij;

            qij = 1.0 / (1.0 + dij_sq);

            double mult = qij * qij;

            thread_sQ += qij;
            for (int ax = 0; ax < n_dimensions; ax++) {
                neg_f[i * n_dimensions + ax] += mult * distance_grad(pos[i*2 + 0], pos[i*2 + 1], pos[j*2 + 0], pos[j*2 + 1], ax);
                //neg_f[i * n_dimensions + ax] = distance_grad(pos[i*2 + 0], pos[i*2 + 1], pos[j*2 + 0], pos[j*2 + 1], ax);
                //neg_f[i * n_dimensions + ax] = mult;
                //neg_f[i * n_dimensions + ax] = distance(0.1, -0.1, 0.3, 0.5);
                //neg_f[i * n_dimensions + ax] = distance_grad(0.1, -0.1, 0.3, 0.5, 0);
            }
        }
    }
    

    atomicAdd(sumQ, thread_sQ);
}